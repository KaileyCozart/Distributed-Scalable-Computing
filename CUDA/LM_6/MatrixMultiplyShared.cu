#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
using namespace std;

#define N 200 // The xy Dimensions/Size of the Two Input Matrices
#define blockSize 32

// MatrixMultiply Kernel
// Assumes N Blocks with 32 Threads Each
__global__ void MatrixMultiply(int *X, int *Y, int *Z, N){

    // Store Elements in Shared Memory Vars
    __shared__ matrixA[N];
    __shared__ matrixB[N];

    // Need to account for different smx tid's
    int Xtid = blockIdx.x * blockDim.x + threadIdx.x;
    int Ytid = blockIdx.y * blockDim.y + threadIdx.y;

    // Copy Matrix from Global to Shared Memory
    for (int i = 0; i < N; i++) {
        matrixA[i] = X[i];
        matrixB[i] = Y[i];
    }

    // Ensure Copy is Complete by Syncing
    __syncthreads();

    // Assuming Square Matrices
    float floatingPointSum = 0.00f;

    // Do the Multiplication
    for (int i = 0; i < N; i++) {

        // Sum Each Corresponding Multiplication, Using Shared Copies of Matrices 
        floatingPointSum = floatingPointSum + (matrixA[(Xtid * N) + i] * matrixB[(i * N) + Ytid]);

    }

    // Sync Again
    __syncthreads();

    // Put the Result in the Output Array
    Z[(Xtid * N) + Ytid] = floatingPointSum;

}

int main(){

    int X[N], Y[N], Z[N]; // Input Data: X, Y; Output Data: Z
    int *dev_X, *dev_Y, *dev_Z; // Device data pointers

    // Allocate Memory on the Device/GPU
    hipMalloc((void**)&dev_X, N*sizeof(int));
    hipMalloc((void**)&dev_Y, N*sizeof(int));
    hipMalloc((void**)&dev_Z, N*sizeof(int));

    // Fill Input Arrays that are Size N x N 
    int arrayLength = N * N;
    for(int i = 0; i < arrayLength; i++){
        X[i] = hiprand_uniform(&localState);
        Y[i] = hiprand_uniform(&localState);
        Z[i] = hiprand_uniform(&localState);
    }

    // Copy data to the device
    hipMemcpy(dev_X,X,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_Y,Y,N*sizeof(int),hipMemcpyHostToDevice);

    // Create Event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Cuda Kernel Call
    int gridSize = (N+(blockSize-1)) / blockSize;

    // Call Event
    hipEventRecord(start);
    MatrixMultiply<<<gridSize,blockSize>>>(dev_X, dev_Y, dev_Z, sN);
    hipEventRecord(stop);

    // Copy memory off of the device
    hipMemcpy(Z, dev_Z, N*sizeof(int), hipMemcpyDeviceToHost);

    // Stop Event
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time Elapsed: " << milliseconds << endl;

    // Check Contents of Working Arrays/Output Data
    int checkValue;
    for (int j = 0; j < arrayLength; j++) {
        for (int i = 0; i < N; i++) { // Loop for Checking Each Value
            checkValue = checkValue + (X[(i * N) + i] * Y[(i * N) + i]);
        }
        if (Z[i] != checkValue) {
            cout << "Mismatch " << i << endl;
        }
    }

    // Free Memory
    hipFree(dev_X);
    hipFree(dev_Y);
    hipFree(dev_Z);
}