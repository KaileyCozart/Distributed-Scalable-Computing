#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
using namespace std;

#define N 200 // The xy Dimensions/Size of the Two Input Matrices
#define blockSize 32

// MatrixMultiply Kernel
// Assumes N Blocks with 32 Threads Each
__global__ void MatrixMultiply(int *X, int *Y, int *Z, N){

    // Need to account for different smx tid's
    int Xtid = blockIdx.x * blockDim.x + threadIdx.x;
    int Ytid = blockIdx.y * blockDim.y + threadIdx.y;

    // Assuming Square Matrices
    float floatingPointSum = 0.00f;

    // Do the Multiplication
    for (int i = 0; i < N; i++) {

        // Sum Each Corresponding Multiplication
        floatingPointSum = floatingPointSum + (X[(Xtid * N) + i] * Y[(i * N) + Ytid]);

    }

    // Put the Result in the Output Array
    Z[(Xtid * N) + Ytid] = floatingPointSum;

}

int main(){

    int X[N], Y[N], Z[N]; // Input Data: X, Y; Output Data: Z
    int *dev_X, *dev_Y, *dev_Z; // Device Data Pointers

    // Allocate Memory on the Device/GPU
    hipMalloc((void**)&dev_X, N*sizeof(int));
    hipMalloc((void**)&dev_Y, N*sizeof(int));
    hipMalloc((void**)&dev_Z, N*sizeof(int));

    // Fill Input Arrays that are Size N x N 
    int arrayLength = N * N;
    for(int i = 0; i < arrayLength; i++){
        X[i] = hiprand_uniform(&localState);
        Y[i] = hiprand_uniform(&localState);
        Z[i] = hiprand_uniform(&localState);
    }

    // Copy Data to the Device
    hipMemcpy(dev_X,X,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_Y,Y,N*sizeof(int),hipMemcpyHostToDevice);

    // Create Event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Cuda Kernel Call
    int gridSize = (N+(blockSize-1)) / blockSize;

    // Call Event
    hipEventRecord(start);
    MatrixMultiply<<<gridSize,blockSize>>>(dev_X, dev_Y, dev_Z, sN);
    hipEventRecord(stop);

    // Copy Memory off of the Device
    hipMemcpy(Z, dev_Z, N*sizeof(int), hipMemcpyDeviceToHost);

    // Stop Event
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time Elapsed: " << milliseconds << endl;

    // Check Contents of Working Arrays/Output Data
    int checkValue;
    for (int j = 0; j < arrayLength; j++) {
        for (int i = 0; i < N; i++) { // Loop for Checking Each Value
            checkValue = checkValue + (X[(i * N) + i] * Y[(i * N) + i]);
        }
        if (Z[i] != checkValue) {
            cout << "Mismatch " << i << endl;
        }
    }

    // Free Memory
    hipFree(dev_X);
    hipFree(dev_Y);
    hipFree(dev_Z);
}