#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
using namespace std;

#define N 200 // The xy Dimensions/Size of the Two Input Matrices
#define blockSize 32

/////////////// MatrixMultiply Kernel NAIVE ///////////////
// Assumes N Blocks with 32 Threads Each
__global__ void MatrixMultiplyNaive(int *A, int *B, int *C, N){

    // Need to account for different smx tid's
    int Atid = blockIdx.x * blockDim.x + threadIdx.x;
    int Btid = blockIdx.y * blockDim.y + threadIdx.y;

    // Assuming Square Matrices
    float floatingPointSum = 0.00f;

    // Do the Multiplication
    for (int i = 0; i < N; i++) {

        // Sum Each Corresponding Multiplication
        floatingPointSum = floatingPointSum + (A[(Atid * N) + i] * B[(i * N) + Btid]);

    }

    // Put the Result in the Output Array
    C[(Atid * N) + Btid] = floatingPointSum;

}

/////////////// MatrixMultiply Kernel SHARED ///////////////
// Assumes N Blocks with 32 Threads Each
__global__ void MatrixMultiplyShared(int *X, int *Y, int *Z, N){

    // Store Elements in Shared Memory Vars
    __shared__ matrixA[N];
    __shared__ matrixB[N];

    // Need to account for different smx tid's
    int Xtid = blockIdx.x * blockDim.x + threadIdx.x;
    int Ytid = blockIdx.y * blockDim.y + threadIdx.y;

    // Copy Matrix from Global to Shared Memory
    for (int i = 0; i < N; i++) {
        matrixA[i] = X[i];
        matrixB[i] = Y[i];
    }

    // Ensure Copy is Complete by Syncing
    __syncthreads();

    // Assuming Square Matrices
    float floatingPointSum = 0.00f;

    // Do the Multiplication
    for (int i = 0; i < N; i++) {

        // Sum Each Corresponding Multiplication, Using Shared Copies of Matrices 
        floatingPointSum = floatingPointSum + (matrixA[(Xtid * N) + i] * matrixB[(i * N) + Ytid]);

    }

    // Sync Again
    __syncthreads();

    // Put the Result in the Output Array
    Z[(Xtid * N) + Ytid] = floatingPointSum;

}

/////////////// MatrixAdd Kernel ///////////////
// Assumes N Blocks with 32 Threads Each
__global__ void MatrixAdd(int *C, int *Z, int *Output){

    // Need to Account for Different SMX Tid's
    int Ctid = blockIdx.x * blockDim.x + threadIdx.x;
    int Ztid = blockIdx.y * blockDim.y + threadIdx.y;

    // Assuming Square Matrices
    float floatingPointSum = 0.00f;

    // Do the Addition
    int maximumXvalue = N
    int location = maximumXvalue * Ztid + Ctid;

    // Put the Result in the Output Array
    if (location < N) {
        Output[location] = C[location] + Z[location];
    }

}

/////////////// Main ///////////////
int main(){

    int A[N * N], B[N * N], C[N * N], X[N * N], Y[N * N], Z[N * N], Output[N * N]; // Input Data: X, Y; Output Data: Z
    int *dev_A, *dev_B, *dev_C, *dev_X, *dev_Y, *dev_Z, *dev_Output; // Device Data Pointers

    // Allocate Memory on the Device/GPU
    hipMalloc((void**)&dev_A, N*sizeof(int));
    hipMalloc((void**)&dev_B, N*sizeof(int));
    hipMalloc((void**)&dev_C, N*sizeof(int));
    hipMalloc((void**)&dev_X, N*sizeof(int));
    hipMalloc((void**)&dev_Y, N*sizeof(int));
    hipMalloc((void**)&dev_Z, N*sizeof(int));
    hipMalloc((void**)&dev_Output, N*sizeof(int));

    // Fill Input Arrays that are Size N x N 
    int arrayLength = N * N;
    for(int i = 0; i < arrayLength; i++){
        A[i] = hiprand_uniform(&localState);
        B[i] = hiprand_uniform(&localState);
        C[i] = hiprand_uniform(&localState);
        X[i] = hiprand_uniform(&localState);
        Y[i] = hiprand_uniform(&localState);
        Z[i] = hiprand_uniform(&localState);
        Output[i] = hiprand_uniform(&localState);
    }

    /////////////// Stream 1 ///////////////

    // Copy Data to the Device
    hipMemcpyAsync(dev_A,A,N * N*sizeof(int),hipMemcpyHostToDevice, 1);
    hipMemcpyAsync(dev_B,B,N * N*sizeof(int),hipMemcpyHostToDevice, 1);

    // Create Event for Timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Cuda Kernel Call
    int gridSize = (N+(blockSize-1)) / blockSize;

    // Call Event
    hipEventRecord(start, stream = 1);
    MatrixMultiplyNaive<<<gridSize,blockSize, 0, 1>>>(dev_A, dev_B, dev_C);
    hipEventRecord(stop);

    // Copy Memory off of the Device
    hipMemcpyAsync(C, dev_C, N * N*sizeof(int), hipMemcpyDeviceToHost, 3);

    // Stop Event
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time Elapsed: " << milliseconds << endl;

    /////////////// Stream 2 ///////////////

    // Copy Data to the Device
    hipMemcpyAsync(dev_X,X,N * N*sizeof(int),hipMemcpyHostToDevice, 2);
    hipMemcpyAsync(dev_Y,Y,N * N*sizeof(int),hipMemcpyHostToDevice, 2);

    // Create Event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Cuda Kernel Call
    int gridSize = (N+(blockSize-1)) / blockSize;

    // Call Event
    hipEventRecord(start, stream = 2);
    MatrixMultiplyShared<<<gridSize,blockSize, 0, 2>>>(dev_X, dev_Y, dev_Z);
    hipEventRecord(stop);

    // Copy Memory off of the Device
    hipMemcpyAsync(Z, dev_Z, N * N*sizeof(int), hipMemcpyDeviceToHost, 3);

    // Stop Event
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time Elapsed: " << milliseconds << endl;

    /////////////// Stream 3 ///////////////

    // Copy Data to the Device
    hipMemcpy(dev_C,C,N * N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_Z,Z,N * N*sizeof(int),hipMemcpyHostToDevice);

    // Create Event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Cuda Kernel Call
    int gridSize = (N+(blockSize-1)) / blockSize;

    // Call Event
    hipEventRecord(start, stream = 3);
    MatrixMultiplyShared<<<gridSize,blockSize, 0, 3>>>(dev_C, dev_Z, dev_Output);
    hipEventRecord(stop);

    // Copy Memory off of the Device
    hipMemcpy(Output, dev_Output, N * N*sizeof(int), hipMemcpyDeviceToHost);

    // Stop Event
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Time Elapsed: " << milliseconds << endl;

    // Free Memory
    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);
    hipFree(dev_X);
    hipFree(dev_Y);
    hipFree(dev_Z);
    hipFree(dev_Output);
}