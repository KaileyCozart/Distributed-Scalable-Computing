
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#define N 65536
#define A 2
#define blockSize 65

// SAXPY Kernel
// Performs A*X+Y
// Assumes single N blocks with 32 threads each
__global__ void saxpy(int *X, int *Y, int *Z){
    // Need to account for different smx tid's
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid<N){
        Z[tid] = A * X[tid] + Y[tid];
    }
}

int main(){

    int X[N], Y[N], Z[N]; // Host data: X,Y input data, Z output data
    int *dev_X, *dev_Y, *dev_Z; // Device data pointers

    // Allocate memory on the device/GPU
    hipMalloc((void**)&dev_X, N*sizeof(int));
    hipMalloc((void**)&dev_Y, N*sizeof(int));
    hipMalloc((void**)&dev_Z, N*sizeof(int));

    // Fill input arrays
    for(int i = 0; i<N; i++){
        X[i] = i;
        Y[i] = i*i;
    }

    // Copy data to the device
    hipMemcpy(dev_X,X,N*sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(dev_Y,Y,N*sizeof(int),hipMemcpyHostToDevice);

    // Create Event
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Cuda Kernel Call
    int gridSize = (N+(blockSize-1)) / blockSize;
    // (N+31) / 32
    // Call Event
    hipEventRecord(start);
    saxpy<<<gridSize,blockSize>>>(dev_X,dev_Y,dev_Z);
    hipEventRecord(stop);

    // Copy memory off of the device
    hipMemcpy(Z,dev_Z,N*sizeof(int),hipMemcpyDeviceToHost);

    // Stop Event
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << milliseconds << endl;

    // Check out contents of working arrays/output data
    for(int i = 0; i<N; i++){
        int checkValue = A * X[i] + Y[i];
        if (Z[i] != checkValue) {
            cout << "Mismatch " << i << endl;
        }
    }

    // Free up memory
    hipFree(dev_X);
    hipFree(dev_Y);
    hipFree(dev_Z);
}